#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// hash function for per-thread random number generation
__device__ unsigned int hash(unsigned int x) {
    x ^= x >> 16;
    x *= 0x85ebca6b;
    x ^= x >> 13;
    x *= 0xc2b2ae35;
    x ^= x >> 16;
    return x;
}

__global__ void copy_stochastic_kernel(float* target, const float* source, int N, unsigned int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) return;

    // Reinterpret the float as int32
    int32_t src_int = __float_as_int(source[idx]);

    // Generate a random 16-bit integer in [0, 65535]
    uint32_t rand16 = hash(idx + seed) & 0xFFFF;

    // Add the random integer to the source
    uint32_t result_int = src_int + rand16;

    // Mask off the lower 16 bits
    result_int &= 0xFFFF0000;

    // Reinterpret as float and store in target
    target[idx] = __int_as_float(result_int);
}

void copy_stochastic_kernel_launcher(torch::Tensor target, torch::Tensor source, unsigned int seed) {
    const int threads = 1024;
    const int blocks = (source.numel() + threads - 1) / threads;
    int N = source.numel();

    copy_stochastic_kernel<<<blocks, threads>>>(
        target.data_ptr<float>(),
        source.data_ptr<float>(),
        N,
        seed);

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("Error in copy_stochastic_kernel: %s\n", hipGetErrorString(err));
    }
}
